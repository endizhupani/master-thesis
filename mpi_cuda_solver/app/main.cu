#include "hip/hip_runtime.h"
#include "cache_helpers.h"
#include "common.h"
//#include "hemi.h"
#include <cstdlib>
#include <iostream>
#include <math.h>
#include <matrix.h>
#include <omp.h>

using namespace std;
#define EPSILON 0.01
#define N_THREADS 8
#define MAX_ITER 5000

int getChunkSize(int arrayCount, int numElPerLine);

int main(int argc,
         char *argv[]) { // matrix size, cpu percentage

  if (argc <= 2) {
    printf("Please enter the matrix size and the percentage of data that "
           "should remain in the CPU.");
  }
  int deviceCount = 0;
#ifdef __HIPCC__

  CUDA_CHECK_RETURN(hipGetDeviceCount(&deviceCount));

  printf("Number of GPUs: %d\n", deviceCount);
#endif // __HIPCC__
  MatrixConfiguration conf = {deviceCount, atoi(argv[1]), atoi(argv[1]),
                              atof(argv[2])};
  printf("GOT HERE\n\n\n");
  pde_solver::Matrix m(conf);
  printf("GOT HERE12\n\n\n");
  m.Init(75, 100, 100, 0, 100, argc, argv);
  printf("GOT HERE13\n\n\n");
  pde_solver::Matrix new_m = m.CloneShell();
  int num_iter = 0;
  float global_diff = 10;
  ExecutionStats stats = {0.0, 0.0, 0.0, 0.0, 0.0, 0, 0};
  printf("GOT HERE1\n\n\n");
  // double tot_loop_time = 0;
  // double calc_start = MPI_Wtime();
  while (global_diff > EPSILON && num_iter < MAX_ITER) {

    // double t = MPI_Wtime();
    m.LocalSweep(new_m, &stats);
    // tot_loop_time += (MPI_Wtime() - t);
    if (num_iter % 4 == 0) {
      global_diff = m.GlobalDifference(&stats);
    }

    pde_solver::Matrix tmp = m;
    m = new_m;
    new_m = tmp;
    num_iter++;
  }
  printf("GOT HERE2\n\n\n");
  stats.print_to_console();
  printf("GOT HERE3\n\n\n");
  // double calc_time = MPI_Wtime() - calc_start;

  // printf("AVG sweep time: %f\n", (tot_loop_time / num_iter));
  // printf("Total Calculaiton time: %f\n", calc_time);

  // m.Synchronize();
  // m.PrintAllPartitions();
  // m.ShowMatrix();

  new_m.Deallocate();

  m.Finalize();
  printf("iter: %d\n", num_iter);
  printf("difference: %f\n", global_diff);
  return 0;
}

int getChunkSize(int arrayCount, int numElPerLine) {
  int num_threads = omp_get_num_threads();

  int chunk_size = ceil((double)arrayCount / num_threads);
  chunk_size += (chunk_size % numElPerLine);

  return chunk_size;
}